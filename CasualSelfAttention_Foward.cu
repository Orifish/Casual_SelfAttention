#include "hip/hip_runtime.h"
#include <THC/THC.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <torch/extension.h>
#include <torch/serialize/tensor.h>

#include <vector>
#include <hipblas.h>


# define THREADS_PER_BLOCK 1024
# define DIVUP(m, n) ((m) / (n) + ((m) % (n) > 0))



__global__ void casualSA_forward_kernel(int class_num,int N,int C, int* class_index,int* index_num, float* Q,float* K,float* V,int* next_index) {
    // 每个线程完成一个
    // 输入格式，Q、K、V为N*C的Tensor，next_index为N*1的tensor，对应每一行的类对应下一个自己类的索引，获得方法是在pytorch中索引==然后左移一格，class_index为N*1的tensor，对应每一行的类
    // output要预先在pytorch中建立好，大小与QKV相同
    int blk_idx = blockIdx.x;
    int thd_idx = threadIdx.x;
    int idx = blk_idx * blockDim.x + thd_idx;
    if (idx >= batch_size * N * C) {
    return;
    } else {
    // 均假设是先横着走再竖着走
    int row = idx/C;
    int col = idx%C;
    int N_class = class_index[row];     // 确认第几行，是哪一类
    int next_pos = next_index[row];     // 确定下一个自己类是哪一行
    for (int i=0;i<N;i++){
        Q[row*C+col]*K[col*C+row];
    }
    } 
}


__global__ void SortCopy_Kernal(float* Sort_Matrix,int N,int C,float* QKV,int* index_num,int* class_index,int* sort_num,int* Orign){
    int blk_idx = blockIdx.x;
    int thd_idx = threadIdx.x;
    int idx = blk_idx * blockDim.x + thd_idx;
    if (idx >= batch_size * N * C) {
    return;
    }else{
        int row = idx/C;
        int true_class = class_index[row];      // 知道这行是什么class
        int start_row = 0;
        for (int i=true_class;i>0;i--){
            start_row += index_num[true_class];
        }       // 定这种类别的初始行
        int row_target = start_row + sort_num[row];       // 得到我应该把这一行给复制到哪一行

        for (int i = 0;i<C;i++){
            Sort_Matrix[row_target*C + i] = QKV[row*C+i]       // copy过去
        }
        Orign[row_target] = row;        // 把原本的位置保存
    }
}


__global__ void Matrix_Mul(float* QKV,int N,int C,float* output){
    int blk_idx = blockIdx.x;
    int thd_idx = threadIdx.x;
    int idx = blk_idx * blockDim.x + thd_idx;
    int row = idx/C;
    int col = idx%C;
    output[row][col] = 0;
    for(int i=0;i<C;i++){
        output[row][col] = QKV[row][i] * QKV[row][i];       // 不同于广义的矩阵乘法，我们是能避免转置的

    }
    
}


void casualSA_kernel_forward_launcher(int class_num,int N,int C, int* class_index,int* index_num, float* QKV,int* next_index,float* output,int* sort_num) {
    hipError_t err;

    dim3 blocks(DIVUP(N * C, THREADS_PER_BLOCK)); // blockIdx.x(col), blockIdx.y(row)
    dim3 threads(THREADS_PER_BLOCK);

    float* Sort_Matrix[N][C];
    int* Orign[N][1];       // 记录原本是哪行的

    hipMallocManaged(&Sort_Matrix, N*C * sizeof(float));
    hipMallocManaged(&Orign, N * sizeof(int));

    SortCopy_Kernal<<<blocks,threads>>>(Sort_Matrix,N,C,QKV,index_num,class_index,sort_num,Orign);        // 注意index_num必须要已经降序排序完了。

    SA_forward_kernel<<<blocks, threads>>>(batch_size, class_num,N,C,class_index,index_num,Q,K,V,next_index,output);
    // hipDeviceSynchronize();  // for using printf in kernel function
    err = hipGetLastError();
    if (hipSuccess != err) {
    fprintf(stderr, "CUDA kernel failed : %s\n", hipGetErrorString(err));
    exit(-1);
    }
}


// 输入N行C列的Tensor Q、K、V
// sort_num是每一个行对应类，在自己类中的排序，在pytorch中给出，可以用[,:这一行]==index来得到。
// index_num是已经降序排序完的索引对应的数量
int casualSA_forward_wrapper(int class_num,int N,int C,at::Tensor class_index_tensor,at::Tensor index_num_tensor, at::Tensor QKV_tensor,at::Tensor Next_Index,at::Tensor output_tensor,at::Tensor sort_num_tensor) {
    int *class_index = level_end_index_tensor.data_ptr<int>();
    int *index_num = input_features_tensor.data_ptr<float>();
    float *QKV = QKV_tensor.data_ptr<float>();
    int *next_index = Next_Index.data_ptr<int>();
    float *output = output_tensor.data_ptr<float>();
    int* sort_num = sort_num_tensor.data_ptr<int>();

    // hipStream_t stream = at::cuda::getCurrentCUDAStream().stream();
    casualSA_kernel_forward_launcher(class_num,N,C,class_index,index_num,QKV,next_index,output,sort_num);
    return 1;
}



